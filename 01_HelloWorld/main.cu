
#include <hip/hip_runtime.h>
#include <iostream>

__global__ 
void cuda_hello()
{
    printf("Hello World from GPU!\n");
}


int main()
{
    printf("Hello World from CPU!\n");

    // call a kernel
    cuda_hello<<<1,1>>>(); 

    /*
	 Synchronize with GPU to wait for printf to finish.
	 Results of printf are buffered and copied back to
	 the CPU for I/O after the kernel has finished.
	*/
    hipDeviceSynchronize();

    return 0;
}

/*
Exercises:
1) Change the message that is printed by the kernel
2) Write a different kernel (different name, different message)
3) Call the different kernels multiple times
*/