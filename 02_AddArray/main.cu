
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <chrono>
using namespace std::chrono;

// Kernel function to add the elements of two arrays
__global__
void add_0(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = index; i < n; i += stride){
    y[i] = x[i] + y[i];
  }
}

/*
Another style for writting kernel function

Note: If grids contain many more blocks (numBlocks) than the limit
of total number of blocks that can be simultaneously executing in a CUDA
device, the runtime system maintains a list of blocks that need to execute
and assigns new blocks to SMs when previously assigned blocks complete execution.
*/
__global__
void add_1(int n, float *x, float *y)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i<n) {
        y[i] = x[i] + y[i];
    }
}

int main(void)
{
    int N = 1<<25;
    float *x, *y;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Get starting timepoint
    auto start = high_resolution_clock::now();

    // Run kernel on 1M elements on the GPU
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    add_0<<<numBlocks, blockSize>>>(N, x, y);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();
    
    // Get ending timepoint
    auto stop = high_resolution_clock::now();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++){
        maxError = fmax(maxError, fabs(y[i]-3.0f));
    }
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);
    
    // Get duration. Substart timepoints to
    // get duration. To cast it to proper unit
    // use duration cast method
    auto duration = duration_cast<microseconds>(stop - start);
    std::cout << "Time taken by function: "
    << duration.count() << " microseconds" << std::endl;
    
    return 0;
}

/*
Exercises:

1) Experiment with printf() inside the kernel. 
Try printing out the values of threadIdx.xand blockIdx.x for some or all of the threads. 
Do they print in sequential order? Why or why not?

2) Print the value of threadIdx.y or threadIdx.z (or blockIdx.y) in the kernel. 
(Likewise for blockDim and gridDim). Why do these exist? 
How do you get them to take on values other than 0 (1 for the dims)?
*/